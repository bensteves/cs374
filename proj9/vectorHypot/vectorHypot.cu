/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 * Extended for use in CS 374 at Calvin College by Joel C. Adams.
 */

/**
 * Vector hypotenuse: C = sqrt(A*A + B*B).
 *
 * This sample is a very basic sample that implements element by element
 * vector hypotenuse calculation. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 *
 * edits By: Ben Steves         -added logic to compute hypotenuse
 * why       Project 9 for CS374
 * where     Calvin University
 * date      11-30-21
 */

#include <stdio.h>
#include <omp.h>
#include <math.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the hypotenuse using vectors A and B into C. 
 * The 3 vectors have the same number of elements numElements.
 */
__global__
void vectorHypot(const float *A, const float *B, float *C, unsigned long numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = sqrt((A[i]*A[i]) + (B[i]*B[i]));
    }
}

void checkErr(hipError_t err, const char* msg) 
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s (error code %d: '%s')!\n", msg, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Host main routine
 */
int main(int argc, char** argv)
{
    double cudaTime = 0.0, totalCudaTime = 0.0,         //cuda runtime
    	   ABcopyTime = 0.0, totalABcopyTime = 0.0,  //host --> device
    	   hypotTime = 0.0, totalHypotTime = 0.0, //computation
    	   CcopyTime = 0.0, totalCcopyTime = 0.0,   //device --> host
    	   seqTime = 0.0, totalSeqTime = 0.0;       //sequential runtime

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    unsigned long numElements = 50000;
    if (argc == 2) {
      numElements = strtoul( argv[1] , 0, 10 );
    }
    size_t size = numElements * sizeof(float);
    printf("[Vector hypotenuse calculation of %lu elements]\n", numElements);

    // Allocate the host input vectors A & B
    float * h_A = (float *)malloc(size);
    float * h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float * h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // 1a. Allocate the device input vectors A & B
    float * d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    checkErr(err, "Failed to allocate device vector A");
    float * d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    checkErr(err, "Failed to allocate device vector B");

    // 1.b. Allocate the device output vector C
    float * d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    checkErr(err, "Failed to allocate device vector C");

    cudaTime = omp_get_wtime();
    ABcopyTime = omp_get_wtime();
    
    // 2. Copy the host input vectors A and B in host memory 
    //     to the device input vectors in device memory
   // printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device vector A from host to device");


    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device vector B from host to device");
    
    totalABcopyTime = omp_get_wtime() - ABcopyTime;

    hypotTime = omp_get_wtime();
    // 3. Launch the Vector Hypot CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
   // printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorHypot<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();
    checkErr(err, "Failed to launch vectorHypot kernel");

    totalHypotTime = omp_get_wtime() - hypotTime;

    CcopyTime = omp_get_wtime();
    
    // 4. Copy the device result vector in device memory
    //     to the host result vector in host memory.
    //printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    checkErr(err, "Failed to copy vector C from device to host");
    
    totalCcopyTime = omp_get_wtime() - CcopyTime; 
    totalCudaTime = omp_get_wtime() - cudaTime;

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(sqrt((h_A[i]*h_A[i]) + (h_B[i]*h_B[i])) - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
   // printf("CUDA test PASSED\n");
//    printf("CUDA time: %lf\n", stopTime-startTime); 

    // Free device global memory
    err = hipFree(d_A);
    checkErr(err, "Failed to free device vector A");

    err = hipFree(d_B);
    checkErr(err, "Failed to free device vector B");

    err = hipFree(d_C);
    checkErr(err, "Failed to free device vector C");


    seqTime = omp_get_wtime(); 
    // repeat the computation sequentially
    for (int i = 0; i < numElements; ++i)
    {
       h_C[i] = sqrt((h_A[i]*h_A[i]) + (h_B[i]*h_B[i]));
    }
    totalSeqTime = omp_get_wtime() - seqTime;

    // verify again
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(sqrt((h_A[i]*h_A[i]) + (h_B[i]*h_B[i])) - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("\nNormal test PASSED\n");
//    printf("Normal time: %lf\n", stopTime-startTime); 
    
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();
    checkErr(err, "Unable to reset device");

    printf("Done\n");

    printf("Cuda runtime: %f \nSequential runtime: %f\n", totalCudaTime, totalSeqTime);
    printf("AB Copy: %f \nHypotenuse: %f \nC copy: %f \n", totalABcopyTime, totalHypotTime, totalCcopyTime);

    return 0;
}

